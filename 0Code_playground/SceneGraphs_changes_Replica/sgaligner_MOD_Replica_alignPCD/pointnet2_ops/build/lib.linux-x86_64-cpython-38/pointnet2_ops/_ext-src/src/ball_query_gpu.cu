#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <torch/extension.h>

#include "cuda_utils.h"

// input: new_xyz(b, m, 3) xyz(b, n, 3)
// output: idx(b, m, nsample)
template<typename scalar_t>
__global__ void query_ball_point_kernel(
    int b,
    int n,
    int m,
    float radius,
    int nsample,
    const scalar_t *__restrict__ new_xyz,
    const scalar_t *__restrict__ xyz,
    int *__restrict__ idx) {
  int bs_idx = blockIdx.y;
  int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (bs_idx >= b || pt_idx >= m) return;

  new_xyz += bs_idx * m * 3 + pt_idx * 3;
  xyz += bs_idx * n * 3;
  idx += bs_idx * m * nsample + pt_idx * nsample;

  float radius2 = radius * radius;
  scalar_t new_x = new_xyz[0];
  scalar_t new_y = new_xyz[1];
  scalar_t new_z = new_xyz[2];

  int cnt = 0;
  for (int k = 0; k < n; ++k) {
      scalar_t x = xyz[k * 3 + 0];
      scalar_t y = xyz[k * 3 + 1];
      scalar_t z = xyz[k * 3 + 2];
      scalar_t d2 = (new_x - x) * (new_x - x) + (new_y - y) * (new_y - y) + (new_z - z) * (new_z - z);
      if (d2 < radius2){
          if (cnt == 0){
              for (int l = 0; l < nsample; ++l) {
                  idx[l] = k;
              }
          }
          idx[cnt] = k;
          ++cnt;
          if (cnt >= nsample) break;
      }
  }
}

at::Tensor query_ball_point_kernel_wrapper(
    int b,
    int n,
    int m,
    float radius,
    int nsample,
    const at::Tensor new_xyz,
    const at::Tensor xyz) {
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  at::Tensor idx = torch::zeros(
      {new_xyz.size(0), new_xyz.size(1), nsample},
      at::device(new_xyz.device()).dtype(at::ScalarType::Int));

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    xyz.scalar_type(), "query_ball_cuda", ([&] {
      query_ball_point_kernel<scalar_t><<<b, opt_n_threads(m), 0, stream>>>(
        b,
        n,
        m,
        radius,
        nsample,
        new_xyz.data_ptr<scalar_t>(),
        xyz.data_ptr<scalar_t>(),
        idx.data_ptr<int>());
  }));
  CUDA_CHECK_ERRORS();
  return idx;
}
